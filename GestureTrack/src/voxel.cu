#include "hip/hip_runtime.h"
#ifndef __VOXEL_CU__
#define __VOXEL_CU__

#include <math.h>
#include <cutil_inline.h>


#define MAX_THREADS_PER_BLOCK 512

//#define EMULATE 

#ifdef EMULATE
#define ATOMIC_ADD(a,v) *a+v
#else
#define ATOMIC_ADD(a,v) atomicAdd(a,v)
#endif

// this const should be defined in CUDA libs but I can't find it

__device__ int getIndex(float val, float minDim, float maxDim, int divs) {
	float divSize = (maxDim - minDim)/divs;
	int result =  (int) (val-minDim)/divSize; // will be negative if less than bounds
	result = (result < divs) ? result : -1; // if greater than upper bound set negative to denote out of bounds
	return result;
} 

__global__ void gpu_calcVoxel_kernel(int pointCnt, float* pixels, float minX, float minY, float minZ, float maxX, float maxY, float maxZ, int divX, int divY, int divZ, float* voxGrid) 
{

#ifdef EMULATE
	printf("------\n");
#endif
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	bool validPoint = ((i > 0) && (i < pointCnt))? true : false;
#ifdef EMULATE	
	printf("%i is a is valid? %i\n", i, validPoint);
#endif

	i *=3; // each point is x,y,z
	
	float px = validPoint ? pixels[i++] : -1;
	float py = validPoint ? pixels[i++] : -1;
	float pz = validPoint ? pixels[i] : -1;
	
#ifdef EMULATE	
		printf("    pt (%f,%f, %f)\n", px,py,pz);
#endif
	
	int vx = getIndex(px, minX, maxX, divX);
	int vy = getIndex(py, minY, maxY, divY);
	int vz = getIndex(pz, minZ, maxZ, divZ);
#ifdef EMULATE	
			printf("   i (%i,%i, %i)\n", vx,vy,vz);
#endif
	
	validPoint = validPoint && (vx >= 0) && (vy >= 0) && (vz >= 0) && (vz > 0);
	
	// if not valid pic a random voxel to "inc" by zero to avoid deadlocking during atomic add
	// not sure if this is really neeeded but it can't hurt
	int vi = validPoint ? (vz * divX * divY) + (vy * divY) + vx : i % (divX*divY*divY);
	float inc = validPoint ? 1.0f : 0.0f;
	ATOMIC_ADD(&voxGrid[vi], inc);	
	
	
}

// we want the order to be back to front, bottom to top
// so we want 0, divy - y, 0 to be first

// Wrapper for the __global__ call that sets up the kernel call
extern "C" void gpu_calcVoxel(int pointCnt, float* pixels, float minX, float minY, float minZ, float maxX, float maxY, float maxZ, int divX, int divY, int divZ, float* voxGrid)
{
	int theadsPerBlock = 256;
	int blocks = (int) ceilf(pointCnt/(float) theadsPerBlock);
	gpu_calcVoxel_kernel <<<blocks,theadsPerBlock>>> (pointCnt, pixels, minX, minY, minZ, maxX, maxY, maxZ, divX, divY, divZ, voxGrid);

};









__global__ void gpu_addScale_kernel(int gridSize, float* d_this, float a, float* d_that, float b)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	#ifdef EMULATE
		printf("%i %i %f %f\n", i, gridSize, a, b);
	#endif
	
	#ifdef EMULATE
		if(i< gridSize) {
			printf("this:%f    that:%f ----> %f\n", d_this[i], d_that[i], ((d_this[i] * a) + (d_that[i] * b)));
						
		}
	#endif
	
	
	if(i < gridSize) 
		d_this[i] = ((d_this[i] * a) + (d_that[i] * b));
		
};


	extern "C" void gpu_addScale(int gridSize, float* d_this, float a, float* d_that, float b)
{
	int theadsPerBlock = 256;
	int blocks = (int) ceilf(gridSize/(float) theadsPerBlock);
	gpu_addScale_kernel <<<blocks,theadsPerBlock>>> ( gridSize, d_this,  a,  d_that,  b);

};

__global__ void gpu_mult_kernel(int gridSize, float* d_this, float* d_that)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
		
	if(i < gridSize) 
		d_this[i] *= d_that[i];
		
};


extern "C" void gpu_mult(int gridSize, float* d_this, float* d_that)
{
	int theadsPerBlock = 256;
	int blocks = (int) ceilf(gridSize/(float) theadsPerBlock);
	gpu_mult_kernel <<<blocks,theadsPerBlock>>> ( gridSize, d_this, d_that);

};


__global__ void gpu_add_kernel(int gridSize, float* d_this, float* d_that)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
		
	if(i < gridSize) 
		d_this[i] += d_that[i];
		
};


extern "C" void gpu_add(int gridSize, float* d_this, float* d_that)
{
	int theadsPerBlock = 256;
	int blocks = (int) ceilf(gridSize/(float) theadsPerBlock);
	gpu_add_kernel <<<blocks,theadsPerBlock>>> ( gridSize, d_this, d_that);

};

__global__ void gpu_sub_kernel(int gridSize, float* d_this, float* d_that)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
		
	if(i < gridSize) 
		d_this[i] -= d_that[i];
		
};

__global__ void gpu_sub_kernel2(int gridSize, float* d_this, float* d_a, float* d_b)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
		
	if(i < gridSize) 
		d_this[i] = d_a[i] - d_b[i];
		
};

extern "C" void gpu_sub(int gridSize, float* d_this, float* d_that)
{
	int theadsPerBlock = 256;
	int blocks = (int) ceilf(gridSize/(float) theadsPerBlock);
	gpu_sub_kernel <<<blocks,theadsPerBlock>>> ( gridSize, d_this, d_that);

};

extern "C" void gpu_sub2(int gridSize, float* d_this, float* d_a, float* d_b)
{
	int theadsPerBlock = 256;
	int blocks = (int) ceilf(gridSize/(float) theadsPerBlock);
	gpu_sub_kernel2 <<<blocks,theadsPerBlock>>> ( gridSize, d_this, d_a, d_b);

};


__global__ void gpu_thresh_kernel(int gridSize, float* d_this, float thresh)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(i < gridSize) 
		d_this[i] = d_this[i] >= thresh ? d_this[i]: 0.0f	;	
};

extern "C" void gpu_thresh(int gridSize, float* d_this, float thresh)
{
	int theadsPerBlock = 256;
	int blocks = (int) ceilf(gridSize/(float) theadsPerBlock);
	gpu_thresh_kernel <<<blocks,theadsPerBlock>>> ( gridSize, d_this, thresh);

};

__global__ void gpu_threshSet_kernel(int gridSize, float* d_this, float thresh, float belowVal, float aboveVal)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(i < gridSize) 
		d_this[i] = d_this[i] >= thresh ? aboveVal : belowVal;	
};

extern "C" void gpu_threshSet(int gridSize, float* d_this, float thresh, float belowVal, float aboveVal)
{
	int theadsPerBlock = 256;
	int blocks = (int) ceilf(gridSize/(float) theadsPerBlock);
	gpu_threshSet_kernel <<<blocks,theadsPerBlock>>> ( gridSize, d_this, thresh, belowVal, aboveVal);

};





__global__ void gpu_scalarMult_kernel(int gridSize, float* d_this, float val)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(i < gridSize) 
		d_this[i] *= val;	
		
};

extern "C" void gpu_scalarMult(int gridSize, float* d_this, float val)
{
	int theadsPerBlock = 256;
	int blocks = (int) ceilf(gridSize/(float) theadsPerBlock);
	gpu_thresh_kernel <<<blocks,theadsPerBlock>>> ( gridSize, d_this, val);

};

__global__ void gpu_incIfOverThresh_kernel(int gridSize, float *d_this, float* d_that, float thresh)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	if(i < gridSize) 
		d_this[i] = (d_that[i] > thresh) ? d_this[i]+1 : 0;	
	
};


extern "C" void gpu_incIfOverThresh(int gridSize, float *d_this, float* d_that, float thresh){
	int theadsPerBlock = 256;
	int blocks = (int) ceilf(gridSize/(float) theadsPerBlock);
	gpu_incIfOverThresh_kernel <<<blocks,theadsPerBlock>>> ( gridSize, d_this, d_that, thresh);

};




#endif

